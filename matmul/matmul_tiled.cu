#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

const int N = 1 << 10;
const int SHMEM_SIZE = 1 << 10;

__global__ void tiled_matmul(int *a, int *b, int*c) {
    // Compute each thread's global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Statically allocated shared memory
    __shared__ int s_a[SHMEM_SIZE];
    __shared__ int s_b[SHMEM_SIZE];

    // Sweep tile across matrix and accumulate
    int result = 0;
    for (int i = 0; i < N; i += blockDim.x) {
        // Load in elements for this tile
        s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
        s_b[threadIdx.y * blockDim.x + threadIdx.x] = b[i * N + threadIdx.y * N + col];

        // Wait for both tiles to be loaded before doing computation
        __syncthreads();

        // Do matrix multiplication on the small matrix
        for (int j = 0; j < blockDim.x; j++) {
            result += s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
        }

        // Wait for all threads to finish using current tiles before loading in new ones
        __syncthreads();
    }

    // Write out this thread's result
    c[row * N + col] = result;
}

// Check result
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int tmp = 0;
            for (int k = 0; k < N; k++) {
                tmp += a[i * N + k] * b[k * N + j];
            }

            assert(tmp == c[i * N + j])
        }
    }
}

int main() {
    constexpr size_t bytes = N * N * sizeof(int);

    // Host vectors
    vector<int> h_a(N * N);
    vector<int> h_b(N * N);
    vector<int> h_c(N * N);

    // Initialize matrices
    generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
    generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

    // Allocate device memory
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy data to the device
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    // Threads per CTA dimension
    int THREADS = 32;

    // Blocks
    int BLOCKS = (int)ceil(N / THREADS);

    // Use dim3 structs for block and grid dimensions
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    // Launch kernel
    tiled_matmul<<<blocks, threads>>>(d_a, d_b, d_c);

    // Copy back to the host
    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    // Check result
    verify_result(h_a, h_b, h_c);

    cout << "COMPLETED SUCCESSFULLY\n";

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}